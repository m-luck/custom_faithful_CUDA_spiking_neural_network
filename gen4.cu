#include "hip/hip_runtime.h"
// SPNET: Spiking neural network with axonal conduction delays and STDP
// Created by Eugene M. Izhikevich, May 17, 2004, San Diego, CA
// Saves spiking data each second in file spikes.dat
// To plot spikes, use MATLAB code: load spikes.dat;plot(spikes(:,1),spikes(:,2),'.');
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "params.h"
#define getrandom(max1) ((rand()%(int)((max1)))) // random integer between 0 and max-1

__global__ void millisecondUpdate(float * devI,
																	int * devpost,
																	float * devs,
																	float * devsd,
																	short * devdelays_length,
																	short * devdelays,
																	int * devN_pre,
																	int * devI_pre,
																	int * devD_pre,
																	float * devs_pre,
																	float * devsd_pre,
																	float * devLTP,
																	float * devLTD,
																	float * deva,
																	float * devd,
																	float * devv,
																	float * devu,
																	int * devfirings,
																	int * devN_firings,
																	int t)
{

	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < devN) {
		if (devv[i]>=30)					// did it fire?
		{
			devv[i] = -65.0;					// voltage reset
			devu[i]+=devd[i];					// recovery variable reset
			devLTP[i*(1000+D)+t+D]= 0.1;
			devLTD[i]=0.12;
			for (int j=0;j<devN_pre[i];j++) devsd_pre[ i*(3*M)+j ] += devLTP[ ( devI_pre [i*(3*M)+j] ) * (1001+D) + t+D-devD_pre[i*3*M+j]-1 ];// this spike was after pre-synaptic spikes
			devfirings[*devN_firings * 2]=t;
			devfirings[*devN_firings++ * 2 + 1]=i;
			if (*devN_firings == devN_firings_max) { printf("Two many spikes at t=%d (ignoring all)",t); *devN_firings=1;}
		}
		int k=*devN_firings;
		while (t-devfirings[(--k)*2] <D)
		{
			for (int j=0; j< devdelays_length[devfirings[k*2+1] * D + t-devfirings[k*2]]; j++)
			{
				i=devpost[ devfirings[k*2+1] * M + devdelays[devfirings[k*2+1] * D + (t-devfirings[k*2] * M + j)]];
				devI[i]+=devs[devfirings[k*2+1] * M + devdelays[devfirings[k*2+1] * D + (t-devfirings[k*0] * M + j)]];
				if (devfirings[k*2+1] <Ne) // this spike is before postsynaptic spikes
					devsd[devfirings[k*2+1] * M + devdelays[devfirings[k*2+1] * D + (t-devfirings[k*2] *M + j)]]-=devLTD[i];
			}
		}
		for (int i=0;i<N;i++)
		{
			devv[i]+=0.5*((0.04*devv[i]+5)*devv[i]+140-devu[i]+devI[i]); // for numerical stability
			devv[i]+=0.5*((0.04*devv[i]+5)*devv[i]+140-devu[i]+devI[i]); // time step is 0.5 ms
			devu[i]+=deva[i]*(0.2*devv[i]-devu[i]);
			devLTP[i * (1001+D) + t+D+1]=0.95*devLTP[i * (1001+D) + t+D];
			devLTD[i]*=0.95;
		}
	}
}

int main()
{
	int		i, j, k, sec, t;
	FILE	*fs;
	initialize(devpost);	// assign connections, weights, etc.

// Neurons take a lot of matrices...time for the most hipMalloc'ing ever done by me.
	hipMalloc(&devpost, sizeof(int)*N*M);
	hipMalloc(&devs, sizeof(int)*N*M);
	hipMalloc(&devsd, sizeof(int)*N*M);
	hipMalloc(&devdelays_length, sizeof(short)*N*D);
	hipMalloc(&devdelays, sizeof(short)*N*D*M);
	hipMalloc(&devN_pre, sizeof(int)*N);
	hipMalloc(&devI_pre, sizeof(int)*N*3*M);
	hipMalloc(&devD_pre, sizeof(int)*N*3*M);
	hipMalloc(&devs_pre, sizeof(float*)*N*3*M);
	hipMalloc(&devsd_pre, sizeof(float*)*N*3*M);
	hipMalloc(&devLTP, sizeof(float)*N*(1001+D));
	hipMalloc(&devLTD, sizeof(float)*N);
	hipMalloc(&deva, sizeof(float)*N);
	hipMalloc(&devd, sizeof(float)*N);
	hipMalloc(&devv, sizeof(float)*N);
	hipMalloc(&devu, sizeof(float)*N);
	hipMalloc(&devfirings, sizeof(int)*N_firings_max*2);

	for (sec=0; sec<60*60*24; sec++)		// simulation of 1 day
	{
		for (t=0;t<1000;t++)				// simulation of 1 sec
		{
			for (i=0;i<N;i++) I[i] = 0.0;	// reset the input
			for (k=0;k<N/1000;k++)
				I[getrandom(N)]=20.0;		// random thalamic input

			hipMemcpy(devN_firings, &N_firings, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(devI, I, sizeof(float)*N, hipMemcpyHostToDevice);
			hipMemcpy(devpost, post, sizeof(int)*N*M, hipMemcpyHostToDevice);
			hipMemcpy(devs, s, sizeof(float)*N*M, hipMemcpyHostToDevice);
			hipMemcpy(devsd, d, sizeof(float)*N*M, hipMemcpyHostToDevice);
			hipMemcpy(devdelays_length, delays_length, sizeof(short)*N*D, hipMemcpyHostToDevice);
			hipMemcpy(devdelays, delays, sizeof(short)*N*D*M, hipMemcpyHostToDevice);
			hipMemcpy(devN_pre, N_pre, sizeof(int)*N, hipMemcpyHostToDevice);
			hipMemcpy(devI_pre, I_pre, sizeof(int)*N*3*M, hipMemcpyHostToDevice);
			hipMemcpy(devD_pre, D_pre, sizeof(int)*N*3*M, hipMemcpyHostToDevice);
			hipMemcpy(devs_pre, s_pre, sizeof(float*)*N*3*M, hipMemcpyHostToDevice);
			hipMemcpy(devsd_pre, sd_pre, sizeof(float*)*N*3*M, hipMemcpyHostToDevice);
			hipMemcpy(devLTP, LTP, sizeof(float)*N*(1001+D), hipMemcpyHostToDevice);
			hipMemcpy(devLTD, LTD, sizeof(float)*N, hipMemcpyHostToDevice);
			hipMemcpy(deva, a, sizeof(float)*N, hipMemcpyHostToDevice);
			hipMemcpy(devd, d, sizeof(float)*N, hipMemcpyHostToDevice);
			hipMemcpy(devv, v, sizeof(float)*N, hipMemcpyHostToDevice);
			hipMemcpy(devu, u, sizeof(float)*N, hipMemcpyHostToDevice);
			hipMemcpy(devfirings, firings, sizeof(int)*N_firings_max*2, hipMemcpyHostToDevice);
			millisecondUpdate<<<7,128>>>(devI,
																	devpost,
																	devs,
																	devsd,
																	devdelays_length,
																	devdelays,
																	devN_pre,
																	devI_pre,
																	devD_pre,
																	devs_pre,
																	devsd_pre,
																	devLTP,
				 													devLTD,
				  												deva,
																	devd,
																	devv,
																	devu,
																	devfirings,
																	devN_firings,
																	t);
			hipMemcpy(devfirings, firings, sizeof(int)*N_firings_max*2, hipMemcpyDeviceToHost);
			hipMemcpy(devu, u, sizeof(float)*N, hipMemcpyDeviceToHost);
			hipMemcpy(v, devv,  sizeof(float)*N, hipMemcpyDeviceToHost);
			hipMemcpy(d, devd, sizeof(float)*N, hipMemcpyDeviceToHost);
			hipMemcpy(a, deva, sizeof(float)*N, hipMemcpyDeviceToHost);
			hipMemcpy(LTD, devLTD, sizeof(float)*N, hipMemcpyDeviceToHost);
			hipMemcpy(LTP, devLTP, sizeof(float)*N*(1001+D), hipMemcpyDeviceToHost);
			hipMemcpy(sd_pre, devsd_pre, sizeof(float*)*N*3*M, hipMemcpyDeviceToHost);
			hipMemcpy(s_pre, devs_pre, sizeof(float*)*N*3*M, hipMemcpyDeviceToHost);
			hipMemcpy(D_pre, devD_pre,  sizeof(int)*N*3*M, hipMemcpyDeviceToHost);
			hipMemcpy(I_pre, devI_pre, sizeof(int)*N*3*M, hipMemcpyDeviceToHost);
			hipMemcpy(N_pre, devN_pre, sizeof(int)*N, hipMemcpyDeviceToHost);
			hipMemcpy(delays, devdelays, sizeof(short)*N*D*M, hipMemcpyDeviceToHost);
			hipMemcpy(delays_length, devdelays_length, sizeof(short)*N*D, hipMemcpyDeviceToHost);
			hipMemcpy(sd, devsd, sizeof(float)*N*M, hipMemcpyDeviceToHost);
			hipMemcpy(s, devs, sizeof(float)*N*M, hipMemcpyDeviceToHost);
			hipMemcpy(post, devpost, sizeof(int)*N*M, hipMemcpyDeviceToHost);
			hipMemcpy(I, devI, sizeof(float)*N, hipMemcpyDeviceToHost);
			hipMemcpy(&N_firings, devN_firings, sizeof(int), hipMemcpyDeviceToHost);
		}
		std::cout << "sec=" << sec << ", firing rate=" << float(N_firings)/N << "\n";
   		fs = fopen("spikes.dat","w");
		for (i=1;i<N_firings;i++)
			if (firings[i][0] >=0)
				fprintf(fs, "%d  %d\n", firings[i][0], firings[i][1]);
		fclose(fs);

		for (i=0;i<N;i++)		// prepare for the next sec
			for (j=0;j<D+1;j++)
			LTP[i][j]=LTP[i][1000+j];
		k=N_firings-1;
		while (1000-firings[k][0]<D) k--;
		for (i=1;i<N_firings-k;i++)
		{
			firings[i][0]=firings[k+i][0]-1000;
			firings[i][1]=firings[k+i][1];
		}
		N_firings = N_firings-k;

		for (i=0;i<Ne;i++)	// modify only exc connections
		for (j=0;j<M;j++)
		{
			s[i][j]+=0.01+sd[i][j];
			sd[i][j]*=0.9;
			if (s[i][j]>sm) s[i][j]=sm;
			if (s[i][j]<0) s[i][j]=0.0;
		}
	}
}
